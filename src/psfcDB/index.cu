#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include ""
#include "index.h"

// CUDA kernel to filter nodes within a given region
__global__ void filterNodes(Node** pages, int* numNodes, double topLatitude, double leftLongitude,
                            double bottomLatitude, double rightLongitude, bool* results) {
    if (threadIdx.x < numNodes[blockIdx.x]) {
        Node* page = pages[blockIdx.x];

        double latitude = page[threadIdx.x].latitude;
        bool latitudeInRange = latitude >= bottomLatitude && latitude <= topLatitude;

        double longitude = page[threadIdx.x].longitude;
        bool longitudeInRange = longitude >= leftLongitude && longitude <= rightLongitude;

        bool inRegion = latitudeInRange && longitudeInRange;

        if (inRegion) {
            results[blockIdx.x * blockDim.x + threadIdx.x] = true;
        } else {
            results[blockIdx.x * blockDim.x + threadIdx.x] = false;
        }
    }
}

PsfcIndex::PsfcIndex() {
    Node* dummyNode = new Node(0, 0, -MAX_LAT, -MAX_LONG, "placeholder");
    MemPage* firstPage = new MemPage(0);
    firstPage->addNode(dummyNode);
    pageList.appendPage(firstPage);
    pageMap[firstPage->startIndex] = firstPage;
    dirtyPages.insert(firstPage);
}

MemPage* PsfcIndex::findPage(const map<uint64_t, MemPage*>& pageMap, uint64_t zIndex) {
    auto it = pageMap.upper_bound(zIndex);
    if (it == pageMap.begin()) {
        return nullptr;
    }
    --it;
    return it->second;
}

void PsfcIndex::addNode(Node* node) {
    MemPage* pageToInsert = findPage(pageMap, node->zIndex);
    if (pageToInsert == nullptr) {
        cerr << "Error finding the first page!" << endl;
    }

    int nodeAdded = pageToInsert->addNode(node);
    if (nodeAdded < 0) {
        MemPage* nextPage = pageToInsert->splitPage();
        if (dirtyPages.count(pageToInsert) > 0) {
            dirtyPages.erase(pageToInsert);
        }

        pageMap[nextPage->startIndex] = nextPage;

        if (node->zIndex < nextPage->startIndex) {
            pageToInsert->addNode(node);
            dirtyPages.insert(pageToInsert);
        } else {
            nextPage->addNode(node);
            dirtyPages.insert(nextPage);
        }
    } else if (nodeAdded == 1) {
        dirtyPages.insert(pageToInsert);
    }
}

void PsfcIndex::flushAll() { pageList.flushAll(); }

// count the nodes in the corners, corner if defined as <lat, long>
int PsfcIndex::findNodes_Host(tuple<double, double> topLeft, tuple<double, double> bottomRight) {
    int result = 0;

    uint64_t start = coordToZIndex(get<0>(topLeft), get<1>(topLeft));
    uint64_t finish = coordToZIndex(get<0>(bottomRight), get<1>(bottomRight));

    int pageVisted = 0;
    MemPage* currPage = findPage(pageMap, start);
    while (currPage != nullptr && currPage->startIndex <= finish) {
        for (int i = 0; i < currPage->size; i++) {
            Node* currNode = &currPage->h_data[i];
            // cout << currNode->to_string() << " ";

            if (currNode->inRegion(topLeft, bottomRight)) {
                result++;
            }
        }

        currPage = currPage->next;
        pageVisted++;
        // cout << endl;
    }

    return result;
}

// count the nodes in the corners, corner if defined as <lat, long>
int PsfcIndex::findNodes_device(tuple<double, double> topLeft, tuple<double, double> bottomRight) {
    int count = 0;

    double topLatitude = std::get<0>(topLeft);
    double leftLongitude = std::get<1>(topLeft);
    double bottomLatitude = std::get<0>(bottomRight);
    double rightLongitude = std::get<1>(bottomRight);

    uint64_t start = coordToZIndex(get<0>(topLeft), get<1>(topLeft));
    uint64_t finish = coordToZIndex(get<0>(bottomRight), get<1>(bottomRight));

    int pageVisted = 0;
    MemPage* currPage = findPage(pageMap, start);

    Node** h_pages;
    Node** d_pages;
    hipHostMalloc((Node***)&h_pages, BATCH_SIZE * sizeof(Node*));
    hipMalloc((Node***)&d_pages, BATCH_SIZE * sizeof(Node*));

    int* h_numNodes;
    int* d_numNodes;
    hipHostMalloc((int**)&h_numNodes, BATCH_SIZE * sizeof(int));
    hipMalloc((int**)&d_numNodes, BATCH_SIZE * sizeof(int));

    bool* d_results;
    hipMalloc((bool**)&d_results, BATCH_SIZE * PAGE_SIZE * sizeof(bool));

    int pageCount = 0;
    while (currPage != nullptr && currPage->startIndex <= finish) {
        h_pages[pageCount] = currPage->d_data;
        h_numNodes[pageCount] = currPage->size;
        pageCount++;

        if (pageCount == BATCH_SIZE) {
            hipMemcpy(d_pages, h_pages, BATCH_SIZE * sizeof(Node*), hipMemcpyHostToDevice);
            hipMemcpy(d_numNodes, h_numNodes, BATCH_SIZE * sizeof(int), hipMemcpyHostToDevice);

            dim3 blockSize(PAGE_SIZE);
            dim3 gridSize(BATCH_SIZE);
            filterNodes<<<gridSize, blockSize>>>(d_pages, d_numNodes, topLatitude, leftLongitude,
                                                 bottomLatitude, rightLongitude, d_results);

            bool* results = new bool[BATCH_SIZE * PAGE_SIZE];

            hipMemcpy(results, d_results, BATCH_SIZE * PAGE_SIZE * sizeof(bool),
                       hipMemcpyDeviceToHost);

            for (int i = 0; i < BATCH_SIZE; i++) {
                for (int j = 0; j < h_numNodes[i]; j++) {
                    if (results[i * PAGE_SIZE + j]) {
                        count++;
                    }
                }
            }
            pageCount = 0;
        }

        currPage = currPage->next;
        pageVisted++;
    }

    if (pageCount != 0) {
        hipMemcpy(d_pages, h_pages, pageCount * sizeof(Node*), hipMemcpyHostToDevice);
        hipMemcpy(d_numNodes, h_numNodes, pageCount * sizeof(int), hipMemcpyHostToDevice);

        dim3 blockSize(PAGE_SIZE);
        dim3 gridSize(pageCount);
        filterNodes<<<gridSize, blockSize>>>(d_pages, d_numNodes, topLatitude, leftLongitude,
                                             bottomLatitude, rightLongitude, d_results);

        bool* results = new bool[pageCount * PAGE_SIZE];

        hipMemcpy(results, d_results, pageCount * PAGE_SIZE * sizeof(bool),
                   hipMemcpyDeviceToHost);

        for (int i = 0; i < pageCount; i++) {
            for (int j = 0; j < h_numNodes[i]; j++) {
                if (results[i * PAGE_SIZE + j]) {
                    count++;
                }
            }
        }
    }

    hipFree(d_pages);
    hipFree(d_numNodes);
    hipFree(d_results);

    cout << "Pages visited: " << pageVisted << endl;
    return count;
}

vector<Node*> PsfcIndex::findNodesNamed_Host(tuple<double, double> topLeft,
                                             tuple<double, double> bottomRight, const char* name) {
    vector<Node*> result;

    uint64_t start = coordToZIndex(get<0>(topLeft), get<1>(topLeft));
    uint64_t finish = coordToZIndex(get<0>(bottomRight), get<1>(bottomRight));

    int pageVisted = 0;
    MemPage* currPage = findPage(pageMap, start);
    while (currPage != nullptr && currPage->startIndex <= finish) {
        for (int i = 0; i < currPage->size; i++) {
            Node* currNode = &currPage->h_data[i];
            if (currNode->zIndex > finish) {
                break;
            }

            if (currNode->inRegion(topLeft, bottomRight) && currNode->nameContains(name)) {
                result.push_back(currNode);
            }
        }
        currPage = currPage->next;
        pageVisted++;
    }

    cout << "Pages visited: " << pageVisted << endl;
    return result;
}

int PsfcIndex::numPages() { return pageMap.size(); }

void PsfcIndex::printIndex() {
    cout << "================== index ===================" << endl;
    pageList.printList();
}
